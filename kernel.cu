#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include ""
#include "time.h" 
#include <stdio.h>
#include "Particles.h"
#include "Turbulences.h"


using namespace std;

#define D 3
#define N 200
#define K 512
#define Nt 20
#define Rt 0.1f
#define c 0.001f
#define ct 0.0001f

__global__ void NextQTur(float* Qt, float* Pt) {
	int i = threadIdx.x;
	Qt[i + 0] += Pt[i + 0] * ct;
	Qt[i + 1] += Pt[i + 1] * ct;
	Qt[i + 2] += Pt[i + 2] * ct;
}

__global__ void Sqrt(float* Q, float* P, float* Qt, float* Pt, float* Eg, float* Epg) {
	int x = blockIdx.x;
	int y = threadIdx.x;
	int i = x * K * D + y * D;
	//int z = threadIdx.z;
	//printf("I = %i \n", x);
	for (int j = 0; j < 3; j++) {
		Q[i + j] = 0.01;
		Qt[i + j] = 0.6;
		P[i + j] = 0.3;
		Pt[i + j] = 0.5;
		Epg[i / D ] = 100000;
		Eg[i / D ] = 0.5;
	}
}

__global__ void addcuda(float* Q, float* P, float* Qt, float* Pt, float* Eg, float* Epg) {
	for (int j = 0; j < 10; j++) {
		int x = blockIdx.x;
		int y = threadIdx.x;
		int i = x * K * D + y * D;

		float Px = P[i + 0];
		float Py = P[i + 1];
		float Pz = P[i + 2];
		float E = Eg[i/3];
		float Ep = Epg[i/3];

		float Qx = Q[i + 0];
		float Qy = Q[i + 1];
		float Qz = Q[i + 2];

		float nQx = Q[i + 0] + c * P[i + 0];
		float nQy = Q[i + 1] + c * P[i + 1];
		float nQz = Q[i + 2] + c * P[i + 2];

		// ��������� �� ������ �������

		if ((nQx > 1) || (nQx < 0)) {
			Px = (-1) * Px;
		}
		if ((nQy > 1) || (nQy < 0)) {
			Py = (-1) * Py;
		}
		if ((nQz > 1) || (nQz < 0)) {
			Pz = (-1) * Pz;
		}

		// ��������� �� ���������������

		for (int nt = 0; nt < Nt; nt += 1) {
			float Range = (sqrt(pow(Qx - Qt[nt + 0], 2) + pow(Qy - Qt[nt + 1], 2) + pow(Qz - Qt[nt + 2], 2)));
			float nRange = (sqrt(pow(nQx - Qt[nt + 0], 2) + pow(nQy - Qt[nt + 1], 2) + pow(nQz - Qt[nt + 2], 2)));

			if((Range > Rt) && (nRange < Rt)) {
				float DirX = (nQx - Qt[nt + 0]) / Range;
				float DirY = (nQy - Qt[nt + 1]) / Range;
				float DirZ = (nQz - Qt[nt + 2]) / Range;
				float PnormKoe = ((Px * DirX) + (Py * DirY) + (Pz * DirZ));
				float Pnormt = ((Pt[nt + 0] * DirX) + (Pt[nt + 1] * DirY) + (Pt[nt + 2] * DirZ));
				E -= (ct / c) * (PnormKoe * PnormKoe) * (Pnormt * abs(Pnormt));
				Px -= 2 * DirX;
				Py -= 2 * DirY;
				Pz -= 2 * DirZ;
			}
		}
		// ������� �������� �� �������, ������������ �� ������� � ������������ �� ���������� ��������. 
		// ������� ���������� ��������� �� ����������
		// Ep ��������� �������� ������� ��������� �� �������
		if ((nQz > 1) && (E > Ep)) {
			E = 100.0f;
		}
		// �������������� ����������
		if (nQz > 0.5) {
			E -= 0.0001f;
		}
		//���������� ������� ��� ����������� ������
		if (((nQz > 0.5f) && (Qz < 0.5f)) || ((Qz > 0.5f) && (nQz < 0.5f))) {
			E += 1.0f;
		}

		// ������ � ������
		Q[i + 0] = nQx;
		Q[i + 1] = nQy;
		Q[i + 2] = nQz;

		P[i + 0] = Px;
		P[i + 1] = Py;
		P[i + 2] = Pz;
		Eg[i/3] = E;
	}
}

int main() {
	// ���������� ��������� �� ����
	FILE* fout;
	// ��������� ���� �� ������
	fout = fopen("DataE.txt", "w");

	float* Q = 0;
	hipMalloc((void**)&Q, N * K * D * sizeof(float));
	float* P = 0;
	hipMalloc((void**)&P, N * K * D * sizeof(float));
	float* Qt = 0;
	hipMalloc((void**)&Qt, N * K * D * sizeof(float));
	float* Pt = 0;
	hipMalloc((void**)&Pt, N * K * D * sizeof(float));
	float* E = 0;
	hipMalloc((void**)&E, N * K * sizeof(float));
	float* Ep = 0;
	hipMalloc((void**)&Ep, N * K * sizeof(float));
	 
	unsigned int start_time = clock();

	

	RandomGenQu(Q, N * K * 3, 0, 9);
	RandomGenPu(P, N * K * 3, 0, 23);
	RandomGenEpu(Ep, N * K, 0, 5, 1000.0f , 10000000.0f);
	ConstEu << <N, K >> > (E);

	RandomGenQ(Qt, Nt * 3, 0, 7);
	RandomGenP(Pt, Nt * 3, 0, 8);
	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// asynchronously issue work to the GPU (all to stream 0)
	hipEventRecord(start, 0);
	//Sqrt <<< N, K >>> (Q, P, Qt, Pt, E, Ep);
	float* f = new float[N * K];
	for (int k = 0; k < 100; k++) {
		hipDeviceSynchronize();
		if (k == 99) {
			hipMemcpy(f, E, N * K * sizeof(float), hipMemcpyDeviceToHost);
			for (int j = 0; j < K * N; j++) {
				fprintf(fout, "%f  ", f[j]);
			}
			fprintf(fout, "\n");
		}
		for (int i = 0; i < 50; i++) {
			addcuda << < N, K >> > (Q, P, Qt, Pt, E, Ep);
			NextQTur << < 1, Nt >> > (Qt, Pt);
		}
	}
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	// print the cpu and gpu times
	printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);

	
	hipDeviceSynchronize();
	unsigned int end_time = clock(); // �������� �����
	unsigned int search_time = end_time - start_time; // ������� �����
	
	
	printf("\n Time %i ms", search_time);

	return 0;
}